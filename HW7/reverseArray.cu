//  reverseArray.cu
//  Chenfeng Hao
//  HW 7
//  Using a second array


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>

using namespace std;

#define ARRAY_SIZE 20
#define BLOCK_SIZE 4


__global__ void cu_reverseArray(int arr_in[], int arr_out[]) {
    // compute thread index
    // use it to retrieve block and thread IDs
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    arr_out[x] = arr_in[ARRAY_SIZE - x - 1];
}


int main(int argc, char *argv[]) {
    // declare arrays and initialize to 0
    int arr_in[ARRAY_SIZE];
    int arr_out[ARRAY_SIZE];

//    srand(0);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        arr_in[i] = i;
    }

    // print initial arrays
    for (int i = 0; i < ARRAY_SIZE; i++) {
        cout << arr_in[i] << ' ';
    }
    cout << endl << endl;

    int *arr_in_d;
    int *arr_out_d;
    hipError_t result;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate space on the device
    result = hipMalloc((void **) &arr_in_d, sizeof(int) * ARRAY_SIZE);
    result = hipMalloc((void **) &arr_out_d, sizeof(int) * ARRAY_SIZE);
    if (result != hipSuccess) {
        cerr << "hipMalloc (thread) failed." << endl;
        exit(1);
    }

    // copy arrays from host to device
    result = hipMemcpy(arr_in_d, arr_in, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);
    result = hipMemcpy(arr_out_d, arr_out, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);
    if (result != hipSuccess) {
        cerr << "hipMemcpy host->dev failed." << endl;
        exit(1);
    }

    // set execution configuration
    dim3 dimblock(BLOCK_SIZE);
    dim3 dimgrid(ARRAY_SIZE / BLOCK_SIZE);

    // call the kernel function
    hipEventRecord(start);
    cu_reverseArray <<<dimgrid, dimblock>>>(arr_in_d, arr_out_d);
    hipEventRecord(stop);

    // transfer results back to host
    result = hipMemcpy(arr_out, arr_out_d, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
        cerr << "hipMemcpy host <- dev (thread) failed." << endl;
        exit(1);
    }

    // release the memory on the GPU
    result = hipFree(arr_out_d);
    result = hipFree(arr_in_d);
    if (result != hipSuccess) {
        cerr << "hipFree (thread) failed." << endl;
        exit(1);
    }

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // re-print arrays
    for (int i = 0; i < ARRAY_SIZE; i++) {
        cout << arr_out[i] << ' ';
    }

    cout << endl << endl;

    cout << "Reversing an array of size " << ARRAY_SIZE << " took " << milliseconds * 1000 << " microseconds." << endl;

    return 0;
}